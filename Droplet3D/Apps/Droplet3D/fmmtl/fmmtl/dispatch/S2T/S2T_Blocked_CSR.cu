#include "hip/hip_runtime.h"
#pragma once

#include <thrust/system/cuda/detail/detail/uninitialized.h>
using thrust::system::cuda::detail::detail::uninitialized_array;

#include "fmmtl/meta/kernel_traits.hpp"

/** CSR Blocked S2T in CUDA
 * @brief  Computes the kernel matrix-vector product using a blocked CSR-like
 *     format. Each target range has a range of source ranges to compute.
 *
 * @param[in] K                 The kernel to generate matrix elements.
 * @param[in] target_range      Maps blockIdx.x to pair<uint,uint> representing
                                 the [start,end) of targets for this threadblock.
 * @param[in] source_range_ptr  Maps blockIdx.x to pair<uint,uint> representing
                                 the [start,end) of the source ranges interacting
                                 the target range for this threadblock.
 * @param[in] source_range      Maps each index of the source_range_ptr range to
                                 a [start,end) of a source range to interact
 *                               with each target of this threadblock..
 *
 * @param[in]     source  Array of sources to index into
 * @param[in]     charge  Array of charges associated with sources to index into
 * @param[in]     target  Array of targets to index into
 * @param[in,out] result  Array of results associated with targets to accumulate
 *
 * @pre For all k, target_range[k].second - target_range[k].first <= blockDim.x
 *       -- One target/thread = each target range is smaller than the blocksize
 */
template <unsigned BLOCKDIM,
          typename Kernel,
          typename Indexable1,
          typename Indexable2,
          typename Indexable3>
__global__ void
blocked_p2p(const Kernel K,
            Indexable1 target_range,
            Indexable2 source_range_ptr,
            Indexable3 source_range,
            const typename KernelTraits<Kernel>::source_type* source,
            const typename KernelTraits<Kernel>::charge_type* charge,
            const typename KernelTraits<Kernel>::target_type* target,
                  typename KernelTraits<Kernel>::result_type* result) {
  typedef typename KernelTraits<Kernel>::source_type source_type;
  typedef typename KernelTraits<Kernel>::charge_type charge_type;
  typedef typename KernelTraits<Kernel>::target_type target_type;
  typedef typename KernelTraits<Kernel>::result_type result_type;

  typedef thrust::pair<unsigned, const unsigned> upair;

  // Allocate shared memory -- prevent initialization of non-POD
  __shared__ uninitialized_array<source_type,BLOCKDIM> sh_s;
  __shared__ uninitialized_array<charge_type,BLOCKDIM> sh_c;

  // Get the target range this block is responsible for
  upair t_range = target_range[blockIdx.x];
  // The target index this thread is responsible for
  t_range.first += threadIdx.x;

  // Get the range of source ranges this block is responsible for
  upair s_range_ptr = source_range_ptr[blockIdx.x];

  // Each thread is assigned to one target in the target range
  result_type r = result_type();
  target_type t = ((t_range.first < t_range.second)
                   ? target[t_range.first] : target_type());

  // For each source range
  for ( ; s_range_ptr.first < s_range_ptr.second; ++s_range_ptr.first) {
    // Get the source range
    upair s_range = source_range[s_range_ptr.first];

    // For each chuck of sources
    for ( ; s_range.first < s_range.second; s_range.first += BLOCKDIM) {

      // Read up to blockDim.x sources into shared memory
      unsigned n = min(s_range.second - s_range.first, BLOCKDIM);
      if (threadIdx.x < n) {
        sh_s[threadIdx.x] = source[s_range.first + threadIdx.x];
        sh_c[threadIdx.x] = charge[s_range.first + threadIdx.x];
      }
      __syncthreads();

      // Each target computes its interaction with each source in smem
      if (t_range.first < t_range.second) {
        do {  // Note that n >= 1 by @pre for (s_range.first ...
          --n;
          r += K(t, sh_s[n]) * sh_c[n];
        } while (n != 0);
      }
      __syncthreads();   // TODO: Unroll to prevent an extra __syncthreads()?
    }
  }

  if (t_range.first < t_range.second)
    result[t_range.first] += r;
}
